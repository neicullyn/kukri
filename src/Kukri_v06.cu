#include "hip/hip_runtime.h"
#include "Kukri.cuh"

using namespace kukri;
#define _BOX_V06 64
#define _BLOCK_SIZE_X_V06 64
#define _BLOCK_SIZE_Y_V06 8

#define _K_LEN 64

#define _STRID_Y_V06 _BLOCK_SIZE_Y_V06
#define _N_LINE_Y_V06 ((_BOX_V06 + _BLOCK_SIZE_Y_V06 - 1) / _BLOCK_SIZE_Y_V06)


void kukri::half_mm_v06(const half *d_A, size_t pitch_A, const half *d_B, size_t pitch_B, half *d_C, int M, int N, int K) {
    dim3 grid_size;
    dim3 block_size;

    block_size.x = _BLOCK_SIZE_X_V06;
    block_size.y = _BLOCK_SIZE_Y_V06;
    block_size.z = 1;

    grid_size.x = M / _BOX_V06;
    grid_size.y = N / _BOX_V06;
    grid_size.z = 1;

    int n_iter = K / _K_LEN;

    //printf("%d %d %d\n", M, N, K);
    if (!((M % _BOX_V06 == 0) && (N % _BOX_V06 == 0) && (K % _K_LEN == 0))) {
        printf("%d %d %d\n", M, N, K);
        printf("Dimension error...\n");
        exit(-1);
    }

    
    kukri::_half_mm_v06_kernel<<<grid_size, block_size>>>(d_A, pitch_A / sizeof(kukri::half), d_B, pitch_B / sizeof(kukri::half), d_C, M, N, K, n_iter);

}
#define _BUF_A_LD (_BOX_V06 + 1)
#define _BUF_B_LD (_K_LEN + 1)
// Assume M, N ,K are all mutiples of _BOX_V06
__global__ void kukri::_half_mm_v06_kernel(const half *d_A, int ld_A, const half *d_B, int ld_B, half *d_C, int M, int N, int K, int n_iter) {
    __shared__ float buf_A[_BUF_A_LD * _K_LEN];
    __shared__ float buf_B[_BUF_B_LD * _BOX_V06];

    int m_offset = _BOX_V06 * blockIdx.x;
    int n_offset = _BOX_V06 * blockIdx.y;

    int x = threadIdx.x;

    float val[_N_LINE_Y_V06];
    int yf[_N_LINE_Y_V06];


    for (int i = 0; i < _N_LINE_Y_V06; i++) {
        val[i] = 0;
    }

    for (int i = 0; i < _N_LINE_Y_V06; i++) {
        yf[i] = threadIdx.y + i * _STRID_Y_V06;
    }


    for (int i_iter = 0; i_iter < n_iter; i_iter++) {
        // Loading the block into shared memory

        int k_offset = _K_LEN * i_iter;


        for (int i = 0; i < _N_LINE_Y_V06; i++) {
            int y = yf[i];
            if (y < _K_LEN) {
                buf_A[IDX2C(x, y, _BUF_A_LD)] = __half2float(d_A[IDX2C(x + m_offset, y + k_offset, ld_A)]);
            }
            if (x < _K_LEN) {
                buf_B[IDX2C(x, y, _BUF_B_LD)] = __half2float(d_B[IDX2C(x + k_offset, y + n_offset, ld_B)]);
            }
        }

        __syncthreads();

        
        for (int k = 0; k < _K_LEN; k++) {
            float a = buf_A[IDX2C(x, k, _BUF_A_LD)];
            for (int i = 0; i < _N_LINE_Y_V06; i++) {
                int y = yf[i];
                float b = buf_B[IDX2C(k, y, _BUF_B_LD)];
                val[i] += a * b;
            }
        }
        __syncthreads();
    }


    for (int i = 0; i < _N_LINE_Y_V06; i++) {
        int y = yf[i];
        d_C[IDX2C(x+m_offset, y+n_offset, M)] = __float2half_rn(val[i]);
    }
}
