#include "hip/hip_runtime.h"
#include "Kukri.cuh"

using namespace kukri;

void kukri::Timer::tic() {
    gpuErrChk(hipEventCreate(&m_start));
    gpuErrChk(hipEventCreate(&m_stop));
    gpuErrChk(hipEventRecord(m_start));
}

float kukri::Timer::toc() {
    gpuErrChk(hipEventRecord(m_stop));
    gpuErrChk(hipEventSynchronize(m_stop));
    gpuErrChk(hipEventElapsedTime(&t, m_start, m_stop));
    gpuErrChk(hipEventDestroy(m_start));
    gpuErrChk(hipEventDestroy(m_stop));
    return t;
}

void kukri::array_half2float_host(float *h_dst, half *h_src, size_t size) {
    // Convert an array of half to an array of float, both of which are in host
    float *d_dst;
    half *d_src;

    gpuErrChk(hipMalloc(&d_dst, size * sizeof(float)));
    gpuErrChk(hipMalloc(&d_src, size * sizeof(half)));

    gpuErrChk(hipMemcpy(d_src, h_src, size * sizeof(half), hipMemcpyHostToDevice));

    kukri::array_half2float_device(d_dst, d_src, size);

    gpuErrChk(hipMemcpy(h_dst, d_dst, size * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_dst);
    hipFree(d_src);
}

void kukri::array_half2float_device(float *d_dst, half *d_src, size_t size) {
    int block_size = 512;
    int grid_size = 256;
    kukri::_array_half2float_kernel <<<grid_size, block_size>>> (d_dst, d_src, size);
    gpuErrChk(hipGetLastError());
}

__global__ void kukri::_array_half2float_kernel(float *d_dst, half *d_src, size_t size) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    size_t step_size = blockDim.x * gridDim.x;
    for (; index < size; index += step_size) {
        d_dst[index] = __half2float(d_src[index]);
    }
}

void kukri::array_float2half_host(half *h_dst, float *h_src, size_t size) {
    // Convert an array of float to an array of half, both of which are in host
    half *d_dst;
    float *d_src;

    gpuErrChk(hipMalloc(&d_dst, size * sizeof(half)));
    gpuErrChk(hipMalloc(&d_src, size * sizeof(float)));

    gpuErrChk(hipMemcpy(d_src, h_src, size * sizeof(float), hipMemcpyHostToDevice));

    kukri::array_float2half_device(d_dst, d_src, size);

    hipMemcpy(h_dst, d_dst, size * sizeof(half), hipMemcpyDeviceToHost);

    gpuErrChk(hipFree(d_dst));
    gpuErrChk(hipFree(d_src));
}

void kukri::array_float2half_device(half *d_dst, float *d_src, size_t size) {
    int block_size = 512;
    int grid_size = 256;

    kukri::_array_float2half_kernel<<<grid_size, block_size>>>(d_dst, d_src, size);
    gpuErrChk(hipGetLastError());
}

__global__ void kukri::_array_float2half_kernel(half *d_dst, float *d_src, size_t size) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    size_t step_size = blockDim.x * gridDim.x;
    for (; index < size; index += step_size) {
        d_dst[index] = __float2half_rn(d_src[index]);
    }
}
